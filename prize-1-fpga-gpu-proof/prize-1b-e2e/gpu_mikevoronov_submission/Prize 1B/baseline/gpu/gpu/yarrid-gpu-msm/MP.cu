#include "hip/hip_runtime.h"
/***

Portions of this file originated in an NVIDIA Open Source Project.  The source can be
found here:  http://github.com/NVlabs/CGBN, from: include/cgbn/arith/mp.cu.

Copyright (c) 2022-2024, Yrrid Software, Inc.  All rights reserved.
Licensed under the Apache License, Version 2.0, see LICENSE for details.

Written by Niall Emmart.

***/

__device__ __constant__ uint32_t zc=0;

__device__ __forceinline__ uint32_t computeNP0(uint32_t x) {
  uint32_t inv=x;

  inv=inv*(inv*x+14);
  inv=inv*(inv*x+2);
  inv=inv*(inv*x+2);
  inv=inv*(inv*x+2);
  return inv;
}

template<uint32_t limbs>
__device__ __forceinline__ void mp_zero(uint32_t* r) {
  #pragma unroll
  for(int32_t i=0;i<limbs;i++)
    r[i]=0;
}

template<uint32_t limbs>
__device__ __forceinline__ void mp_one(uint32_t* r) {
  r[0]=1;
  #pragma unroll
  for(int32_t i=1;i<limbs;i++)
    r[i]=0;
}

template<uint32_t limbs>
__device__ __forceinline__ void mp_copy(uint32_t* dst, const uint32_t* src) {
  #pragma unroll
  for(int32_t i=0;i<limbs;i++)
    dst[i]=src[i];
}

template<uint32_t limbs>
__device__ __forceinline__ uint32_t mp_logical_or(const uint32_t* a) {
  uint32_t lor=a[0];
  
  #pragma unroll
  for(int32_t i=1;i<limbs;i++)
    lor=lor | a[i];
  return lor;
}

template<uint32_t limbs>
__device__ __forceinline__ uint32_t mp_shift_right(uint32_t* r, const uint32_t* x, uint32_t bits, uint32_t fill=0) {
  #pragma unroll
  for(int32_t i=0;i<limbs-1;i++)
    r[i]=__funnelshift_rc(x[i], x[i+1], bits);
  r[limbs-1]=__funnelshift_rc(x[limbs-1], fill, bits);
}

template<uint32_t limbs>
__device__ __forceinline__ uint32_t mp_shift_left(uint32_t* r, const uint32_t* x, uint32_t bits, uint32_t fill=0) {
  #pragma unroll
  for(int32_t i=limbs-1;i>0;i--)
    r[i]=__funnelshift_lc(x[i-1], x[i], bits);
  r[0]=__funnelshift_lc(fill, x[0], bits);
}

template<uint32_t limbs>
__device__ __forceinline__ void mp_add(uint32_t* r, const uint32_t* a, const uint32_t* b) {
  chain_t chain;
  
  #pragma unroll
  for(int32_t i=0;i<limbs;i++)
    r[i]=chain.add(a[i], b[i]);
}

template<uint32_t limbs>
__device__ __forceinline__ bool mp_add_carry(uint32_t* r, const uint32_t* a, const uint32_t* b) {
  chain_t chain;
  
  #pragma unroll
  for(int32_t i=0;i<limbs;i++)
    r[i]=chain.add(a[i], b[i]);
  return chain.getCarry();
}

template<uint32_t limbs>
__device__ __forceinline__ void mp_sub(uint32_t* r, const uint32_t* a, const uint32_t* b) {
  chain_t chain;
  
  #pragma unroll
  for(int32_t i=0;i<limbs;i++)
    r[i]=chain.sub(a[i], b[i]);
}

template<uint32_t limbs>
__device__ __forceinline__ bool mp_sub_carry(uint32_t* r, const uint32_t* a, const uint32_t* b) {
  chain_t chain;
  
  #pragma unroll
  for(int32_t i=0;i<limbs;i++)
    r[i]=chain.sub(a[i], b[i]);
  return chain.getCarry();
}

template<uint32_t limbs>
__device__ __forceinline__ void mp_neg(uint32_t* r, const uint32_t* a) {
  chain_t chain;

  #pragma unroll
  for(int32_t i=0;i<limbs;i++) 
    r[i]=chain.sub(0, a[i]);
}

template<uint32_t limbs>
__device__ __forceinline__ bool mp_neg_carry(uint32_t* r, const uint32_t* a) {
  chain_t chain;

  #pragma unroll
  for(int32_t i=0;i<limbs;i++) 
    r[i]=chain.sub(0, a[i]);
  return chain.getCarry();
}

template<uint32_t limbs>
__device__ __forceinline__ bool mp_comp_eq(const uint32_t* a, const uint32_t* b) {
  uint32_t match=a[0] ^ b[0];
  
  #pragma unroll
  for(int32_t i=1;i<limbs;i++)
    match=match | (a[i] ^ b[i]);
  return match==0;
}

template<uint32_t limbs>
__device__ __forceinline__ bool mp_comp_ge(const uint32_t* a, const uint32_t* b) {
  chain_t chain;
  
  #pragma unroll
  for(int32_t i=0;i<limbs;i++)
    chain.sub(a[i], b[i]);
  return chain.getCarry();
}

template<uint32_t limbs>
__device__ __forceinline__ bool mp_comp_gt(const uint32_t* a, const uint32_t* b) {
  chain_t chain;
  
  // a>b --> b-a is negative
  #pragma unroll
  for(int32_t i=0;i<limbs;i++)
    chain.sub(b[i], a[i]);
  return !chain.getCarry();
}

template<uint32_t limbs>
__device__ __forceinline__ void mp_swap(uint32_t* a, uint32_t* b) {
  uint32_t swap;

  #pragma unroll
  for(uint32_t i=0;i<limbs;i++) {
    swap=a[i];
    a[i]=b[i];
    b[i]=swap;
  }
}

template<uint32_t limbs>
__device__ __forceinline__ void mp_select(uint32_t* r, bool abSelect, const uint32_t* a, const uint32_t* b) {
  #pragma unroll
  for(int32_t i=0;i<limbs;i++) 
    r[i]=abSelect ? a[i] : b[i];
}

template<uint32_t limbs>
__device__ __forceinline__ void mp_scale(uint32_t* r, const uint32_t* a, const uint32_t scalar) {
  uint64_t wide=0;

  // this is not very fast, don't use it for performance critical computations
  #pragma unroll
  for(int32_t i=0;i<limbs;i++) {
    wide=madwide(a[i], scalar, wide>>32);
    r[i]=ulow(wide);
  }
}

template<class NP, uint32_t limbs>
__device__ __forceinline__ bool mp_mul_red_cl(uint64_t* evenOdd, const uint32_t* a, const uint32_t* b, const uint32_t* n) {
  uint64_t* even=evenOdd;
  uint64_t* odd=evenOdd + limbs/2;
  chain_t   chain;
  bool      carry=false;
  uint32_t  lo=0, q, c1, c2;
  
  // This routine can be used when max(a, b)+n < R (i.e. it doesn't carry out).  Hence the name cl for carryless.
  // Only works with an even number of limbs.
     
  #pragma unroll
  for(int32_t i=0;i<limbs/2;i++) {
    even[i]=make_wide(0, 0);
    odd[i]=make_wide(0, 0);
  }
  
  #pragma unroll
  for(int32_t i=0;i<limbs;i+=2) {
    if(i!=0) {
      // integrate lo
      chain.reset(carry);
      lo=chain.add(lo, ulow(even[0]));
      carry=chain.add(0, 0)!=0;
      even[0]=make_wide(lo, uhigh(even[0]));
    }

    chain.reset();
    #pragma unroll
    for(int j=0;j<limbs;j+=2)
      even[j/2]=chain.madwide(a[i], b[j], even[j/2]);
    c1=chain.add(0, 0);

    chain.reset();
    #pragma unroll
    for(int j=0;j<limbs;j+=2)
      odd[j/2]=chain.madwide(a[i], b[j+1], odd[j/2]);

    q=NP::qTerm(ulow(even[0]));

    chain.reset();
    #pragma unroll
    for(int j=0;j<limbs;j+=2)
      odd[j/2]=chain.madwide(q, n[j+1], odd[j/2]);

    chain.reset();
    even[0]=chain.madwide(q, n[0], even[0]);
    lo=uhigh(even[0]);
    #pragma unroll
    for(int j=2;j<limbs;j+=2)
      even[j/2-1]=chain.madwide(q, n[j], even[j/2]);
    c1=chain.add(c1, 0);
      
    // integrate lo
    
    chain.reset(carry);
    lo=chain.add(lo, ulow(odd[0]));
    carry=chain.add(0, 0)!=0;
    odd[0]=make_wide(lo, uhigh(odd[0]));

    chain.reset();
    #pragma unroll
    for(int j=0;j<limbs;j+=2)
      odd[j/2]=chain.madwide(a[i+1], b[j], odd[j/2]);
    c2=chain.add(0, 0);

    q=NP::qTerm(ulow(odd[0]));

    // shift odd by 64 bits

    chain.reset();
    odd[0]=chain.madwide(q, n[0], odd[0]);
    lo=uhigh(odd[0]);
    #pragma unroll
    for(int j=2;j<limbs;j+=2)
      odd[j/2-1]=chain.madwide(q, n[j], odd[j/2]);
    c2=chain.add(c2, 0);

    odd[limbs/2-1]=make_wide(0, 0);
    even[limbs/2-1]=make_wide(c1, c2);
    
    chain.reset();
    #pragma unroll
    for(int j=0;j<limbs;j+=2)
      even[j/2]=chain.madwide(a[i+1], b[j+1], even[j/2]);

    chain.reset();
    #pragma unroll
    for(int j=0;j<limbs;j+=2)
      even[j/2]=chain.madwide(q, n[j+1], even[j/2]);
  }

  chain.reset(carry);
  lo=chain.add(lo, ulow(even[0]));
  carry=chain.add(0, 0)!=0;
  even[0]=make_wide(lo, uhigh(even[0]));
  return carry;
}

template<class NP, uint32_t limbs>
__device__ __forceinline__ bool mp_sqr_red_cl(uint64_t* evenOdd, uint32_t* temp, const uint32_t* a, const uint32_t* n) {
  uint64_t* even=evenOdd;
  uint64_t* odd=evenOdd + limbs/2;
  chain_t   chain;
  bool      carry=false;
  uint32_t  lo=0, q, c1, c2, low, high;
  
  // This routine can be used when a+n < R (i.e. it doesn't carry out).  Hence the name cl for carryless.
  // Only works with an even number of limbs.
  
  mp_zero<limbs>(temp);
  
  #pragma unroll
  for(int32_t i=0;i<limbs/2;i++) {
    even[i]=make_wide(0, 0);
    odd[i]=make_wide(0, 0);
  }
  
  // do odds
  for(int32_t j=limbs-1;j>0;j-=2) {
    chain.reset();
    for(int i=0;i<limbs-j;i++)
      evenOdd[j/2+i+1]=chain.madwide(a[i], a[i+j], evenOdd[j/2+i+1]);
  }

  // shift right
  for(int32_t i=0;i<limbs-1;i++)
    evenOdd[i]=make_wide(uhigh(evenOdd[i]), ulow(evenOdd[i+1]));
  evenOdd[limbs-1]=make_wide(uhigh(evenOdd[limbs-1]), 0);
   
  // do evens
  for(int32_t j=limbs-2;j>0;j-=2) {
    chain.reset();
    for(int i=0;i<limbs-j;i++) 
      evenOdd[j/2+i]=chain.madwide(a[i], a[i+j], evenOdd[j/2+i]);
    temp[limbs-j]=(chain.add(0, 0)!=0) ? 2 : 0;
  }

  // double
  chain.reset();
  for(int32_t i=0;i<limbs;i++) {
    low=chain.add(ulow(evenOdd[i]), ulow(evenOdd[i]));
    high=chain.add(uhigh(evenOdd[i]), uhigh(evenOdd[i]));
    evenOdd[i]=make_wide(low, high);
  }

  // add diagonals
  chain.reset();
  for(int32_t i=0;i<limbs;i++) 
    evenOdd[i]=chain.madwide(a[i], a[i], evenOdd[i]);

  // add high part of wide to b...
  chain.reset();
  for(int32_t i=0;i<limbs;i+=2) {
    temp[i]=chain.add(ulow(evenOdd[limbs/2+i/2]), temp[i]);
    temp[i+1]=chain.add(uhigh(evenOdd[limbs/2+i/2]), temp[i+1]);
  }

  #pragma unroll
  for(int32_t i=0;i<limbs/2;i++) 
    odd[i]=make_wide(0, 0);

  // now we need to reduce
  #pragma unroll
  for(int i=0;i<limbs/2;i++) {
    if(i!=0) {
      // integrate lo
      chain.reset(carry);
      lo=chain.add(lo, ulow(even[0]));
      carry=chain.add(0, 0)!=0;
      even[0]=make_wide(lo, uhigh(even[0]));
    }
    
    q=NP::qTerm(ulow(even[0]));

    // shift even by 64 bits
    chain.reset();
    even[0]=chain.madwide(q, n[0], even[0]);
    lo=uhigh(even[0]);
    #pragma unroll
    for(int j=2;j<limbs;j+=2)
      even[j/2-1]=chain.madwide(q, n[j], even[j/2]);
    c1=chain.add(0, 0);

    chain.reset();
    #pragma unroll
    for(int j=0;j<limbs;j+=2)
      odd[j/2]=chain.madwide(q, n[j+1], odd[j/2]);
      
    // second half

    // integrate lo
    chain.reset(carry);
    lo=chain.add(lo, ulow(odd[0]));
    carry=chain.add(0, 0)!=0;
    odd[0]=make_wide(lo, uhigh(odd[0]));
    
    q=NP::qTerm(ulow(odd[0]));

    // shift odd by 64 bits
    chain.reset();
    odd[0]=chain.madwide(q, n[0], odd[0]);
    lo=uhigh(odd[0]);
    for(int j=2;j<limbs;j+=2)
      odd[j/2-1]=chain.madwide(q, n[j], odd[j/2]);
    odd[limbs/2-1]=0;
    c2=chain.add(0, 0);

    chain.reset();
    for(int j=0;j<limbs-2;j+=2)
      even[j/2]=chain.madwide(q, n[j+1], even[j/2]);
    even[limbs/2-1]=chain.madwide(q, n[limbs-1], make_wide(c1, c2));
  }
  
  chain.reset();
  for(int i=0;i<limbs;i+=2) {
    low=chain.add(ulow(even[i/2]), temp[i]);
    high=chain.add(uhigh(even[i/2]), temp[i+1]);
    even[i/2]=make_wide(low, high);
  }
  
  chain.reset(carry);
  lo=chain.add(lo, ulow(even[0]));
  carry=chain.add(0, 0)!=0;
  even[0]=make_wide(lo, uhigh(even[0]));
  return carry;
}

template<uint32_t limbs>
__device__ __forceinline__ void mp_merge_cl(uint32_t* r, const uint64_t* evenOdd, bool carry) {
  chain_t chain(carry);
 
  r[0]=ulow(evenOdd[0]);
  for(int i=0;i<limbs/2-1;i++) {
    r[2*i+1]=chain.add(uhigh(evenOdd[i]), ulow(evenOdd[limbs/2 + i]));
    r[2*i+2]=chain.add(ulow(evenOdd[i+1]), uhigh(evenOdd[limbs/2 + i]));
  }
  r[limbs-1]=chain.add(uhigh(evenOdd[limbs/2-1]), 0);
}

template<uint32_t limbs>
__device__ __forceinline__ void mp_inverse(uint32_t* r, const uint32_t* f, const uint32_t* n) {
  uint32_t A[limbs], B[limbs], U[limbs], V[limbs];

  // we assume that n is prime
  mp_copy<limbs>(A, f);
  mp_copy<limbs>(B, n);
  mp_one<limbs>(U);
  mp_zero<limbs>(V);
  
  while(mp_logical_or<limbs>(A)!=0) {
    if((A[0] & 0x01)!=0) {
      if(mp_comp_gt<12>(B, A)) {
        mp_swap<limbs>(A, B);
        mp_swap<limbs>(U, V);
      }
      mp_sub<limbs>(A, A, B);
      if(!mp_sub_carry<limbs>(U, U, V))
        mp_add<limbs>(U, U, n);
    }
    mp_shift_right<limbs>(A, A, 1);
    if((U[0] & 0x01)!=0)
      mp_add<limbs>(U, U, n);
    mp_shift_right<limbs>(U, U, 1);
  }
  mp_copy<limbs>(r, V);
}
