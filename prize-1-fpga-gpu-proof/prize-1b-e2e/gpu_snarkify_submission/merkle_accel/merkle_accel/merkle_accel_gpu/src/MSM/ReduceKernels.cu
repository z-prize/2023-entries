#include "hip/hip_runtime.h"
/***

Copyright (c) 2023-2024, Yrrid Software, Inc. and Snarkify Inc.  All rights reserved.
Dual licensed under the MIT License or the Apache License, Version 2.0.
See LICENSE for details.

Author(s):  Niall Emmart

***/

template<class Curve, uint32_t bucketBits>
__launch_bounds__(256,1)
__global__ void reduceExtendedJacobianBuckets(void* reduceMemory, void* bucketMemory) {
  const uint32_t BUCKET_COUNT=1<<bucketBits;  

  uint32_t globalTID=blockIdx.x*blockDim.x+threadIdx.x, warpThread=threadIdx.x & 0x1F;
  uint32_t bucketsPerThread=(BUCKET_COUNT+gridDim.x*blockDim.x-1)/(gridDim.x*blockDim.x);
  uint32_t loadIndex, storeIndex, scale;
  bool     infinity, bit;

  typename Curve::PointXYZZ sum, sumOfSums, pt;

  Curve::initializeXYZZ(sum);
  Curve::initializeXYZZ(sumOfSums);
  Curve::initializeXYZZ(pt);

  Curve::initializeShared();

  for(int32_t i=bucketsPerThread-1;i>=0;i--) {
    loadIndex=globalTID*bucketsPerThread + i;
    if(loadIndex<BUCKET_COUNT) {
      Curve::load(pt.x, fieldOffset(bucketMemory, loadIndex*FIELDS_XYZZ + FIELD_X));
      Curve::load(pt.y, fieldOffset(bucketMemory, loadIndex*FIELDS_XYZZ + FIELD_Y));
      Curve::load(pt.zz, fieldOffset(bucketMemory, loadIndex*FIELDS_XYZZ + FIELD_ZZ));
      Curve::load(pt.zzz, fieldOffset(bucketMemory, loadIndex*FIELDS_XYZZ + FIELD_ZZZ));
      Curve::accumulateXYZZ(sum, pt);
    }
    Curve::accumulateXYZZ(sumOfSums, sum);
  }

  scale=globalTID * bucketsPerThread; 
  
  // compute sum=scale*sum
  Curve::setXYZZ(pt, sum);
  infinity=true;
  for(int32_t i=31;i>=0;i--) {
    if(!infinity) 
      Curve::doubleXYZZ(sum, sum);
    bit=(scale & (1<<i))!=0;
    if(infinity && bit) 
      infinity=false;
    else if(!infinity && bit)
      Curve::accumulateXYZZ(sum, pt);

    __syncwarp(0xFFFFFFFF);
  }

  if(infinity)
    Curve::setXYZZ(sum, sumOfSums);
  else
    Curve::accumulateXYZZ(sum, sumOfSums);

  // sum across the warp
  #pragma unroll 1
  for(int32_t i=1;i<32;i=i+i) {
    Curve::warpShuffleXYZZ(pt, sum, threadIdx.x ^ i);
    Curve::accumulateXYZZ(sum, pt);
  }

  // since all threads in the warp have the same sum, we can use a trick!
  storeIndex=globalTID>>5;
  if(warpThread==0) {
    Curve::store(fieldOffset(reduceMemory, storeIndex*FIELDS_XYZZ + FIELD_X), sum.x);
    Curve::store(fieldOffset(reduceMemory, storeIndex*FIELDS_XYZZ + FIELD_Y), sum.y);
    Curve::store(fieldOffset(reduceMemory, storeIndex*FIELDS_XYZZ + FIELD_ZZ), sum.zz);
    Curve::store(fieldOffset(reduceMemory, storeIndex*FIELDS_XYZZ + FIELD_ZZZ), sum.zzz);
  }
}

template<class Curve, uint32_t bucketBits>
__launch_bounds__(256,1)
__global__ void reduceTwistedEdwardsBuckets(void* reduceMemory, void* bucketMemory) {
  const uint32_t BUCKET_COUNT=1<<bucketBits;  

  uint32_t globalTID=blockIdx.x*blockDim.x+threadIdx.x, warpThread=threadIdx.x & 0x1F;
  uint32_t bucketsPerThread=(BUCKET_COUNT+gridDim.x*blockDim.x-1)/(gridDim.x*blockDim.x);
  uint32_t loadIndex, storeIndex, scale;
  bool     infinity, bit;

  typename Curve::PointXYTZ sum, sumOfSums, pt;

  Curve::initializeXYTZ(sum);
  Curve::initializeXYTZ(sumOfSums);
  Curve::initializeXYTZ(pt);

  for(int32_t i=bucketsPerThread-1;i>=0;i--) {
    loadIndex=globalTID*bucketsPerThread + i;
    if(loadIndex<BUCKET_COUNT) {
      Curve::load(pt.x, fieldOffset(bucketMemory, loadIndex*FIELDS_XYTZ + FIELD_X));
      Curve::load(pt.y, fieldOffset(bucketMemory, loadIndex*FIELDS_XYTZ + FIELD_Y));
      Curve::load(pt.t, fieldOffset(bucketMemory, loadIndex*FIELDS_XYTZ + FIELD_T));
      Curve::load(pt.z, fieldOffset(bucketMemory, loadIndex*FIELDS_XYTZ + FIELD_Z));
      Curve::accumulateXYTZ(sum, pt);
    }
    Curve::accumulateXYTZ(sumOfSums, sum);
  }

  scale=globalTID * bucketsPerThread; 
  
  // compute sum=scale*sum
  Curve::setXYTZ(pt, sum);
  infinity=true;
  for(int32_t i=31;i>=0;i--) {
    if(!infinity) 
      Curve::accumulateXYTZ(sum, sum);
    bit=(scale & (1<<i))!=0;
    if(infinity && bit) 
      infinity=false;
    else if(!infinity && bit)
      Curve::accumulateXYTZ(sum, pt);

    __syncwarp(0xFFFFFFFF);
  }

  if(infinity)
    Curve::setXYTZ(sum, sumOfSums);
  else
    Curve::accumulateXYTZ(sum, sumOfSums);

  // sum across the warp
  #pragma unroll 1
  for(int32_t i=1;i<32;i=i+i) {
    Curve::warpShuffleXYTZ(pt, sum, threadIdx.x ^ i);
    Curve::accumulateXYTZ(sum, pt);
  }

  // since all threads in the warp have the same sum, we can use a trick!
  storeIndex=globalTID>>5;
  if(warpThread==0) {
    Curve::store(fieldOffset(reduceMemory, storeIndex*FIELDS_XYTZ + FIELD_X), sum.x);
    Curve::store(fieldOffset(reduceMemory, storeIndex*FIELDS_XYTZ + FIELD_Y), sum.y);
    Curve::store(fieldOffset(reduceMemory, storeIndex*FIELDS_XYTZ + FIELD_T), sum.t);
    Curve::store(fieldOffset(reduceMemory, storeIndex*FIELDS_XYTZ + FIELD_Z), sum.z);
  }
}

template<class Curve>
__launch_bounds__(256,1)
__global__ void sumExtendedJacobian(void* result, void* uniformResult, void* reduceMemory, uint32_t pointCount) {
  uint32_t                  warpThread=threadIdx.x & 0x1F, warp=threadIdx.x>>5;
  uint32_t                  sharedBase=1536;  // leave space for tables, total is 1536 + 192*8 = 3K
  typename Curve::PointXYZZ sum, pt;

  Curve::initializeShared();

  Curve::initializeXYZZ(sum);
  if(uniformResult!=NULL && threadIdx.x==0) {
    Curve::load(sum.x, fieldOffset(uniformResult, FIELD_X));
    Curve::load(sum.y, fieldOffset(uniformResult, FIELD_Y));
    Curve::load(sum.zz, fieldOffset(uniformResult, FIELD_ZZ));
    Curve::load(sum.zzz, fieldOffset(uniformResult, FIELD_ZZZ));
    Curve::negateXYZZ(sum);
  }

  __syncthreads();

  for(int32_t idx=threadIdx.x;idx<pointCount;idx+=blockDim.x) {
    Curve::load(pt.x, fieldOffset(reduceMemory, idx*FIELDS_XYZZ + FIELD_X));
    Curve::load(pt.y, fieldOffset(reduceMemory, idx*FIELDS_XYZZ + FIELD_Y));
    Curve::load(pt.zz, fieldOffset(reduceMemory, idx*FIELDS_XYZZ + FIELD_ZZ));
    Curve::load(pt.zzz, fieldOffset(reduceMemory, idx*FIELDS_XYZZ + FIELD_ZZZ));
    Curve::accumulateXYZZ(sum, pt);
  }

  __syncthreads();

  // sum across the warp
  #pragma unroll 1
  for(int32_t i=1;i<32;i=i+i) {
    Curve::warpShuffleXYZZ(pt, sum, threadIdx.x ^ i);
    Curve::accumulateXYZZ(sum, pt);
  }

  if(warpThread==0) {
    Curve::storeShared(sharedBase + warp*48 + 0, sum.x);
    Curve::storeShared(sharedBase + warp*48 + 384, sum.y);
    Curve::storeShared(sharedBase + warp*48 + 768, sum.zz);
    Curve::storeShared(sharedBase + warp*48 + 1152, sum.zzz);
  }

  __syncthreads();

  if(warp==0) {
    Curve::loadShared(sum.x, sharedBase + (warpThread & 0x07)*48 + 0);
    Curve::loadShared(sum.y, sharedBase + (warpThread & 0x07)*48 + 384);
    Curve::loadShared(sum.zz, sharedBase + (warpThread & 0x07)*48 + 768);
    Curve::loadShared(sum.zzz, sharedBase + (warpThread & 0x07)*48 + 1152);

    #pragma unroll 1
    for(int32_t i=1;i<8;i=i+i) {
      Curve::warpShuffleXYZZ(pt, sum, threadIdx.x ^ i);
      Curve::accumulateXYZZ(sum, pt);
    }
  }

  if(threadIdx.x==0) {
    Curve::store(fieldOffset(result, FIELD_X), sum.x);
    Curve::store(fieldOffset(result, FIELD_Y), sum.y);
    Curve::store(fieldOffset(result, FIELD_ZZ), sum.zz);
    Curve::store(fieldOffset(result, FIELD_ZZZ), sum.zzz);
  }    
}

template<class Curve>
__launch_bounds__(256,1)
__global__ void sumAndNormalizeExtendedJacobian(void* result, void* uniformResult, void* reduceMemory, uint32_t pointCount) {
  uint32_t                  warpThread=threadIdx.x & 0x1F, warp=threadIdx.x>>5;
  uint32_t                  sharedBase=1536;  // leave space for tables, total is 1536 + 192*8 = 3K
  typename Curve::PointXYZZ sum, pt;
  typename Curve::PointXY   res;

  // launch a CTA for each MSM in the batch

  Curve::initializeShared();

  Curve::initializeXYZZ(sum);
  if(uniformResult!=NULL && threadIdx.x==0) {
    Curve::load(sum.x, fieldOffset(uniformResult, FIELD_X));
    Curve::load(sum.y, fieldOffset(uniformResult, FIELD_Y));
    Curve::load(sum.zz, fieldOffset(uniformResult, FIELD_ZZ));
    Curve::load(sum.zzz, fieldOffset(uniformResult, FIELD_ZZZ));
    Curve::negateXYZZ(sum);
  }

  __syncthreads();

  reduceMemory=fieldOffset(reduceMemory, blockIdx.x*pointCount*FIELDS_XYZZ);
  result=fieldOffset(result, blockIdx.x*FIELDS_XY);

  for(int32_t idx=threadIdx.x;idx<pointCount;idx+=blockDim.x) {
    Curve::load(pt.x, fieldOffset(reduceMemory, idx*FIELDS_XYZZ + FIELD_X));
    Curve::load(pt.y, fieldOffset(reduceMemory, idx*FIELDS_XYZZ + FIELD_Y));
    Curve::load(pt.zz, fieldOffset(reduceMemory, idx*FIELDS_XYZZ + FIELD_ZZ));
    Curve::load(pt.zzz, fieldOffset(reduceMemory, idx*FIELDS_XYZZ + FIELD_ZZZ));
    Curve::accumulateXYZZ(sum, pt);
  }

  __syncthreads();

  // sum across the warp
  #pragma unroll 1
  for(int32_t i=1;i<32;i=i+i) {
    Curve::warpShuffleXYZZ(pt, sum, threadIdx.x ^ i);
    Curve::accumulateXYZZ(sum, pt);
  }

  if(warpThread==0) {
    Curve::storeShared(sharedBase + warp*48 + 0, sum.x);
    Curve::storeShared(sharedBase + warp*48 + 384, sum.y);
    Curve::storeShared(sharedBase + warp*48 + 768, sum.zz);
    Curve::storeShared(sharedBase + warp*48 + 1152, sum.zzz);
  }

  __syncthreads();

  if(warp==0) {
    Curve::loadShared(sum.x, sharedBase + (warpThread & 0x07)*48 + 0);
    Curve::loadShared(sum.y, sharedBase + (warpThread & 0x07)*48 + 384);
    Curve::loadShared(sum.zz, sharedBase + (warpThread & 0x07)*48 + 768);
    Curve::loadShared(sum.zzz, sharedBase + (warpThread & 0x07)*48 + 1152);

    #pragma unroll 1
    for(int32_t i=1;i<8;i=i+i) {
      Curve::warpShuffleXYZZ(pt, sum, threadIdx.x ^ i);
      Curve::accumulateXYZZ(sum, pt);
    }

    Curve::normalize(res, sum);

    Curve::fromMontgomery(res.x, res.x);
    Curve::fromMontgomery(res.y, res.y);

    Curve::reduceFully(res.x, res.x);
    Curve::reduceFully(res.y, res.y);

    if(threadIdx.x==0) {
      Curve::store(fieldOffset(result, FIELD_X), res.x);
      Curve::store(fieldOffset(result, FIELD_Y), res.y);
    }
  }
}

