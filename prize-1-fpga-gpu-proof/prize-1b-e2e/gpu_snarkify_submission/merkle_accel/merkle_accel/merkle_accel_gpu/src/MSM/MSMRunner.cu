#include "hip/hip_runtime.h"
/***

Copyright (c) 2023-2024, Yrrid Software, Inc. and Snarkify Inc.  All rights reserved.
Dual licensed under the MIT License or the Apache License, Version 2.0.
See LICENSE for details.

Author(s):  Niall Emmart

***/

/*************************************************************************************************
 *
 * Typical memory usage for 2^24 points:
 *
 * total bytes:  N * WINDOWS * (SAFETY + 3) * 4 = 264 * N --> 4224 MB
 *               BUCKET_COUNT * 6 * 4                     --> 96 MB
 *               BINS_PER_GROUP * (GROUP_COUNT + 1) * 4   --> 2.25 MB
 *               Misc crap < 0.5 MB
 *
 *               Scalars:                                       512 MB
 *               Bucket storage:                                384 MB
 *               Point Storage:                               18432 MB
 *                                                         ------------
 *               Total:                                      23,651 MB
 *
 *************************************************************************************************/

void* advancePointer(void*& ptr, size_t bytes) {
  size_t   add=(bytes + 0x3FFL) & 0xFFFFFFFFFFFFFC00L;
  uint8_t* ptr8=(uint8_t*)ptr;
  void*    prior=ptr;

  ptr=(void*)(ptr8+add);
  return prior;
}

void advanceOffset(size_t& current, size_t bytes) {
  size_t add=(bytes + 0x3FFL) & 0xFFFFFFFFFFFFFC00L;

  current=current + add;
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::MSMRunner(uint32_t maxPointCount) {
  _maxPointCount=maxPointCount;
  _uniformBucketPointCount=0xFFFFFFFF;
  _uniformBucketMSM=NULL;
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::~MSMRunner() {
  if(_uniformBucketMSM!=NULL)
    hipFree(_uniformBucketMSM);
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
size_t MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::pointBytesRequired() {
  const uint32_t WINDOWS=Planning::WINDOWS;
  const uint32_t FIELDS_PER_POINT=(accumulation==ACCUMULATION_TWISTED_EDWARDS_XYT) ? 3 : 2;

  return ((size_t)(WINDOWS*FIELDS_PER_POINT*Curve::limbs*4))*_maxPointCount;
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
size_t MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::planningBytesRequired() {
  const uint32_t WINDOWS=Planning::WINDOWS;
  const uint32_t INDEX_BITS=Planning::INDEX_BITS;
  const uint32_t BINS_PER_GROUP=Planning::BINS_PER_GROUP;
  const uint32_t BUCKET_COUNT=Planning::BUCKET_COUNT;

  size_t         current=0;
  uint32_t       groupCount;

  groupCount=_maxPointCount>>INDEX_BITS;
  groupCount=(groupCount>0) ? groupCount : 1;

  advanceOffset(current, _maxPointCount*WINDOWS*4);             // pointIndexes
  advanceOffset(current, BUCKET_COUNT*4);                       // sortedBucketIndexes
  advanceOffset(current, BUCKET_COUNT*4);                       // sortedBucketCounts
  advanceOffset(current, BUCKET_COUNT*4);                       // sortedBucketOffsets

  if(uniformBuckets)
    advanceOffset(current, _maxPointCount*32);                  // randomScalars

  advanceOffset(current, BINS_PER_GROUP*groupCount*4);          // binCounts
  advanceOffset(current, BINS_PER_GROUP*4);                     // binOffsets
  advanceOffset(current, 256*4);                                // bigBinCounts
  advanceOffset(current, _maxPointCount*safety*WINDOWS*4);      // binnedPointIndexes

  advanceOffset(current, 4);                                    // overflowCount
  advanceOffset(current, BUCKET_COUNT*4);                       // bucketOverflowCounts
  advanceOffset(current, _maxPointCount*WINDOWS*8);             // overflowValues

  advanceOffset(current, BUCKET_COUNT*4);                       // bucketCounts
  advanceOffset(current, BUCKET_COUNT*4);                       // bucketOffsets
  advanceOffset(current, 256*4);                                // bucketSizeCounts
  advanceOffset(current, 256*4);                                // bucketSizeNexts

  return current;
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
size_t MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::bucketBytesRequired() {
  const uint32_t BUCKET_BITS=Planning::BUCKET_BITS;
  const uint32_t FIELDS_PER_BUCKET=(accumulation==ACCUMULATION_AFFINE) ? 2 : 4;
  
  return ((size_t)FIELDS_PER_BUCKET*Curve::limbs*4)<<BUCKET_BITS;
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
size_t MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::reduceBytesRequired() {
  const uint32_t MAX_SM_COUNT=256;
  const uint32_t MAX_WARP_COUNT=8;
  const uint32_t FIELDS_PER_BUCKET=4;

  return (size_t)(FIELDS_PER_BUCKET*Curve::limbs*4*MAX_SM_COUNT*MAX_WARP_COUNT); 
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
int32_t MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::
runPointGeneration(hipStream_t stream, void* points, void* secretScalars, uint32_t pointCount) {
  uint32_t* cpuScalars;
  void*     gpuScalars;
  int32_t   ec, smCount;

  if(secretScalars==NULL)
    cpuScalars=(uint32_t*)malloc(pointCount*32);
  else
    cpuScalars=(uint32_t*)secretScalars;

  for(int i=0;i<pointCount;i++) {
    for(int j=0;j<8;j++) 
      cpuScalars[i*8+j]=((rand() & 0xFFFF)<<16) + (rand() & 0xFFFF);
  }

  ec=hipDeviceGetAttribute(&smCount, hipDeviceAttributeMultiprocessorCount, 0);
  if(ec!=hipSuccess) return ec;
  ec=hipMalloc(&gpuScalars, pointCount*32);
  if(ec!=hipSuccess) return ec;
  ec=hipMemcpy(gpuScalars, cpuScalars, pointCount*32, hipMemcpyHostToDevice);
  if(ec!=hipSuccess) return ec;
  generatePoints<Curve><<<smCount, 256, 1536, stream>>>(points, gpuScalars, pointCount);
  ec=hipStreamSynchronize(stream);
  if(ec!=hipSuccess) return ec;
  ec=hipFree(gpuScalars);
  if(ec!=hipSuccess) return ec;
  if(secretScalars==NULL)
    free(cpuScalars);
  return hipSuccess;
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
int32_t MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::
runPointPrecompute(hipStream_t stream, void* scaledPoints, void* sourcePoints, uint32_t pointCount) {
  int32_t ec=0, smCount;

  ec=hipDeviceGetAttribute(&smCount, hipDeviceAttributeMultiprocessorCount, 0);
  if(ec!=hipSuccess) return ec;
  if constexpr (accumulation==ACCUMULATION_AFFINE || accumulation==ACCUMULATION_EXTENDED_JACOBIAN || accumulation==ACCUMULATION_EXTENDED_JACOBIAN_ML)
    scalePoints<Curve, windowBits><<<smCount, 256, 1536, stream>>>(scaledPoints, sourcePoints, pointCount, _maxPointCount);
  else if constexpr (accumulation==ACCUMULATION_TWISTED_EDWARDS_XY) 
    scaleTwistedEdwardsXYPoints<Curve, windowBits><<<smCount, 256, 1536, stream>>>(scaledPoints, sourcePoints, pointCount, _maxPointCount);
  else if constexpr (accumulation==ACCUMULATION_TWISTED_EDWARDS_XYT) 
    scaleTwistedEdwardsXYTPoints<Curve, windowBits><<<smCount, 256, 1536, stream>>>(scaledPoints, sourcePoints, pointCount, _maxPointCount);
  ec=hipStreamSynchronize(stream);
  if(ec!=hipSuccess) return ec;
  return 0;
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
int32_t MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::
runUniformBucketsSetup(hipStream_t stream, void* reduceMemory, void* bucketMemory, void* planningMemory, void* pointMemory, uint32_t pointCount) {
  const uint32_t FIELDS_PER_BUCKET=4;
  const uint32_t WINDOWS=Planning::WINDOWS;
  const uint32_t BUCKET_COUNT=Planning::BUCKET_COUNT;
  const uint32_t EXPONENT_HIGH_WORD=Planning::EXPONENT_HIGH_WORD;

  void*    randomScalars;
  uint32_t resultPointCount;
  size_t   reduceBytes=reduceBytesRequired();
  int32_t  ec;

  // UNIFORM BUCKET SUPPORT:
  // 1.  If this is the first call, or if the point size has changed since the prior call:
  // 2.    If this is the first call then:
  // 3.      Generate random scalars and copy them to the GPU
  // 4.    compute the MSM of the random scalars of size "pointCount"

  if(!uniformBuckets || _uniformBucketPointCount==pointCount)
    return hipSuccess;

  printf("Running uniform setup code\n");

  randomScalars=planningMemory;
  advancePointer(randomScalars, _maxPointCount*WINDOWS*4);
  advancePointer(randomScalars, BUCKET_COUNT*4);
  advancePointer(randomScalars, BUCKET_COUNT*4);
  advancePointer(randomScalars, BUCKET_COUNT*4);

  if(_uniformBucketPointCount==0xFFFFFFFF) {
    uint32_t* cpuRandomScalars=(uint32_t*)malloc(_maxPointCount*32);

    // For a production system, this should be a cryptographic random number generated, with a 128 bit seed.
    // Also, note, this is not quite a uniform distribution, but it's pretty close.  Again, for production,
    // I'd go with a good quality uniform field random generator, rather than just modding the top word.

    for(int i=0;i<_maxPointCount;i++) {
      for(int j=0;j<8;j++) 
        cpuRandomScalars[i*8+j]=((rand() & 0xFFFF)<<16) + (rand() & 0xFFFF);
      cpuRandomScalars[i*8+7]=cpuRandomScalars[i*8+7] % EXPONENT_HIGH_WORD;
    }
    ec=hipMemcpy(randomScalars, cpuRandomScalars, _maxPointCount*32, hipMemcpyHostToDevice);
    if(ec!=hipSuccess)
      return ec;
    free(cpuRandomScalars);
  }

  _uniformBucketPointCount=0xFFFFFFFF;

  ec=hipMalloc(&_uniformBucketMSM, FIELDS_PER_BUCKET*Curve::limbs*4);
  if(ec!=hipSuccess) return ec;
  // now we compute the MSM for the number of points that we have
  ec=runPlanning(stream, planningMemory, NULL, pointCount);
  if(ec!=hipSuccess) return ec;
  ec=runAccumulate(stream, bucketMemory, planningMemory, pointMemory);
  if(ec!=hipSuccess) return ec;
  ec=runReduce(stream, &resultPointCount, reduceMemory, bucketMemory);
  if(ec!=hipSuccess) return ec;
  ec=runFinalReduce(stream, _uniformBucketMSM, reduceMemory, resultPointCount);
  if(ec!=hipSuccess) return ec;
  _uniformBucketPointCount=pointCount;

  return hipSuccess;
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
int32_t MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::
runPlanning(hipStream_t stream, void* planningMemory, void* scalars, uint32_t startPoint, uint32_t stopPoint) {
  const uint32_t WINDOWS=Planning::WINDOWS;
  const uint32_t INDEX_BITS=Planning::INDEX_BITS;
  const uint32_t SEGMENT_BITS=Planning::SEGMENT_BITS;
  const uint32_t BINS_PER_GROUP=Planning::BINS_PER_GROUP;
  const uint32_t BUCKET_COUNT=Planning::BUCKET_COUNT;

  PlanningLayout planning;
  uint32_t       groupBits, groupCount, pointsPerBinGroup;
  int32_t        ec, smCount;

  groupCount=_maxPointCount>>INDEX_BITS;
  groupCount=(groupCount>0) ? groupCount : 1;
  pointsPerBinGroup=safety*WINDOWS<<((INDEX_BITS-SEGMENT_BITS>0) ? INDEX_BITS-SEGMENT_BITS : 0);

  groupBits=0;
  for(int i=groupCount-1;i>0;i=i>>1)
    groupBits++;

  planning.maxPointCount=_maxPointCount;
  planning.groupBits=groupBits;
  planning.groupCount=groupCount;
  planning.pointsPerBinGroup=pointsPerBinGroup;
  planning.binCount=BINS_PER_GROUP*groupCount;

  planning.pointIndexes=(uint32_t*)advancePointer(planningMemory, _maxPointCount*WINDOWS*4);
  planning.sortedBucketIndexes=(uint32_t*)advancePointer(planningMemory, BUCKET_COUNT*4);
  planning.sortedBucketCounts=(uint32_t*)advancePointer(planningMemory, BUCKET_COUNT*4);
  planning.sortedBucketOffsets=(uint32_t*)advancePointer(planningMemory, BUCKET_COUNT*4);

  planning.randomScalars=NULL;
  if(uniformBuckets) 
    planning.randomScalars=(uint32_t*)advancePointer(planningMemory, _maxPointCount*32);

  planning.binCounts=(uint32_t*)advancePointer(planningMemory, BINS_PER_GROUP*groupCount*4);
  planning.binOffsets=(uint32_t*)advancePointer(planningMemory, BINS_PER_GROUP*4);
  planning.bigBinCounts=(uint32_t*)advancePointer(planningMemory, 256*4);
  planning.binnedPointIndexes=(uint32_t*)advancePointer(planningMemory, _maxPointCount*safety*WINDOWS*4);

  planning.overflowCount=(uint32_t*)advancePointer(planningMemory, 4);
  planning.bucketOverflowCounts=(uint32_t*)advancePointer(planningMemory, BUCKET_COUNT*4);
  planning.overflow=(uint2*)advancePointer(planningMemory, _maxPointCount*WINDOWS*8);

  planning.bucketCounts=(uint32_t*)advancePointer(planningMemory, BUCKET_COUNT*4);
  planning.bucketOffsets=(uint32_t*)advancePointer(planningMemory, BUCKET_COUNT*4);
  planning.bucketSizeCounts=(uint32_t*)advancePointer(planningMemory, 256*4);
  planning.bucketSizeNexts=(uint32_t*)advancePointer(planningMemory, 256*4);

  ec=hipDeviceGetAttribute(&smCount, hipDeviceAttributeMultiprocessorCount, 0);
  if(ec!=0) return ec;

  zeroCounters<Planning><<<256, 256>>>(planning);
  if constexpr (!uniformBuckets)
    partitionIntoBins<Planning><<<smCount, 512>>>(planning, (uint4*)scalars, startPoint, stopPoint);
  else {
    if(scalars==NULL) 
      partitionIntoBins<Planning><<<smCount, 512>>>(planning, (uint4*)planning.randomScalars, startPoint, stopPoint);
    else 
      partitionIntoBins<Planning><<<smCount, 512>>>(planning, (uint4*)scalars, (uint4*)planning.randomScalars, startPoint, stopPoint);
  }
  computeBinOffsets<Planning><<<256, 256>>>(planning);
  sortBins<Planning><<<BINS_PER_GROUP/64, 256, pointsPerBinGroup*groupCount*4>>>(planning);
  processOverflows<Planning><<<smCount, 512>>>(planning);
  sortByCounts<Planning><<<smCount, 512>>>(planning);

  return 0;
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
int32_t MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::runPlanning(hipStream_t stream, void* planningMemory, void* scalars, uint32_t pointCount) {
  return runPlanning(stream, planningMemory, scalars, 0, pointCount);
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
int32_t MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::
runAccumulate(hipStream_t stream, void* bucketMemory, void* planningMemory, void* pointMemory, bool preloaded) {
  const uint32_t WINDOWS=Planning::WINDOWS;
  const uint32_t BUCKET_COUNT=Planning::BUCKET_COUNT;

  PlanningLayout planning;
  int32_t        ec;

  planning.pointIndexes=(uint32_t*)advancePointer(planningMemory, _maxPointCount*WINDOWS*4);
  planning.sortedBucketIndexes=(uint32_t*)advancePointer(planningMemory, BUCKET_COUNT*4);
  planning.sortedBucketCounts=(uint32_t*)advancePointer(planningMemory, BUCKET_COUNT*4);
  planning.sortedBucketOffsets=(uint32_t*)advancePointer(planningMemory, BUCKET_COUNT*4);

  if constexpr (accumulation==ACCUMULATION_TWISTED_EDWARDS_XYT) {
    ec=hipFuncSetAttribute(reinterpret_cast<const void*>(accumulateTwistedEdwardsXYTBuckets<Curve), 2>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
    if(ec!=0) return ec;
  }

  // about ACCUMULATION_EXTENDED_JACOBIAN and ACCUMULATION_EXTENDED_JACOBIAN_ML are about 1.5% faster with 3 blocks instead of 2
  if constexpr (accumulation==ACCUMULATION_EXTENDED_JACOBIAN) 
    accumulateExtendedJacobianBuckets<Curve, 3><<<BUCKET_COUNT/128, 128, 19616, stream>>>(bucketMemory, planning, pointMemory, preloaded);
  else if constexpr (accumulation==ACCUMULATION_EXTENDED_JACOBIAN_ML) {
    accumulateExtendedJacobianMLBuckets<Curve, 3><<<BUCKET_COUNT/128, 128, 19616, stream>>>(bucketMemory, planning, pointMemory, preloaded);
  }
  else if constexpr (accumulation==ACCUMULATION_TWISTED_EDWARDS_XY) {
    accumulateTwistedEdwardsXYBuckets<Curve, 2><<<BUCKET_COUNT/128, 128, 32416, stream>>>(bucketMemory, planning, pointMemory, preloaded);
  }
  else if constexpr (accumulation==ACCUMULATION_TWISTED_EDWARDS_XYT) {
    accumulateTwistedEdwardsXYTBuckets<Curve, 2><<<BUCKET_COUNT/128, 128, 32416, stream>>>(bucketMemory, planning, pointMemory, preloaded);
  }
  return hipSuccess;
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
int32_t MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::
runReduce(hipStream_t stream, uint32_t* reducePointCount, void* reduceMemory, void* bucketMemory) {
  int32_t ec, smCount;

  if(reducePointCount!=NULL)
    *reducePointCount=0;

  ec=hipDeviceGetAttribute(&smCount, hipDeviceAttributeMultiprocessorCount, 0);
  if(ec!=0) return ec;

  if(reducePointCount!=NULL)
    *reducePointCount=smCount*8;

  if constexpr (accumulation==ACCUMULATION_EXTENDED_JACOBIAN || accumulation==ACCUMULATION_EXTENDED_JACOBIAN_ML) {
    reduceExtendedJacobianBuckets<Curve, windowBits-1><<<smCount, 256, 1536, stream>>>(reduceMemory, bucketMemory);
  }
  else if constexpr (accumulation==ACCUMULATION_TWISTED_EDWARDS_XY || accumulation==ACCUMULATION_TWISTED_EDWARDS_XYT) {
    reduceTwistedEdwardsBuckets<Curve, windowBits-1><<<smCount, 256, 0, stream>>>(reduceMemory, bucketMemory);
  }
  return hipSuccess;
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
int32_t MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::
runFinalReduce(hipStream_t stream, void* finalResult, void* reducedBuckets, uint32_t reducePointCount) {
  if constexpr (accumulation==ACCUMULATION_EXTENDED_JACOBIAN || accumulation==ACCUMULATION_EXTENDED_JACOBIAN_ML) {
    sumExtendedJacobian<Curve><<<1, 256, 3072, stream>>>(finalResult, _uniformBucketPointCount==0xFFFFFFFF ? NULL : _uniformBucketMSM, reducedBuckets, reducePointCount);
  }
  else if constexpr (accumulation==ACCUMULATION_TWISTED_EDWARDS_XY || accumulation==ACCUMULATION_TWISTED_EDWARDS_XYT) {
    printf("ACK!! IMPLEMENT ME!\n");
  }
  return hipSuccess;
}

template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
int32_t MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::
runFinalReduceAndNormalize(hipStream_t stream, void* finalResult, void* reducedBuckets, uint32_t reducePointCount, uint32_t batchCount) {
  if constexpr (accumulation==ACCUMULATION_EXTENDED_JACOBIAN || accumulation==ACCUMULATION_EXTENDED_JACOBIAN_ML) {
    sumAndNormalizeExtendedJacobian<Curve><<<batchCount, 256, 3072, stream>>>(finalResult, _uniformBucketPointCount==0xFFFFFFFF ? NULL : _uniformBucketMSM, reducedBuckets, reducePointCount);
  }
  else if constexpr (accumulation==ACCUMULATION_TWISTED_EDWARDS_XY || accumulation==ACCUMULATION_TWISTED_EDWARDS_XYT) {
    printf("ACK!! IMPLEMENT ME!\n");
  }
  return hipSuccess;
}


template<class Curve, Accumulation accumulation, uint32_t windowBits, uint32_t binBits, uint32_t safety, bool uniformBuckets>
int32_t MSMRunner<Curve, accumulation, windowBits, binBits, safety, uniformBuckets>::dumpOperations(const char* filename, void* planningMemory, uint32_t pointCount) {
  const uint32_t BUCKET_COUNT=Planning::BUCKET_COUNT, WINDOWS=Planning::WINDOWS;

  FILE*     f;
  uint32_t* operations;
  int32_t   ec;
  size_t    bytes=_maxPointCount;     
  uint32_t* pointIndexes;
  uint32_t* sortedBucketIndexes;
  uint32_t* sortedBucketCounts;
  uint32_t* sortedBucketOffsets;
  void*     current;

  f=fopen(filename, "w");
  bytes=bytes*WINDOWS*4 + BUCKET_COUNT*12;

  operations=(uint32_t*)malloc(bytes);
  ec=hipMemcpy(operations, planningMemory, bytes, hipMemcpyDeviceToHost);
  if(ec!=hipSuccess) return ec;

  current=(void*)operations;
  pointIndexes=(uint32_t*)advancePointer(current, _maxPointCount*WINDOWS*4);
  sortedBucketIndexes=(uint32_t*)advancePointer(current, BUCKET_COUNT*4); 
  sortedBucketCounts=(uint32_t*)advancePointer(current, BUCKET_COUNT*4); 
  sortedBucketOffsets=(uint32_t*)advancePointer(current, BUCKET_COUNT*4);

  for(int i=0;i<BUCKET_COUNT;i++) {
    if(sortedBucketCounts[i]+sortedBucketOffsets[i]>=pointCount*WINDOWS) {
      fprintf(stderr, "Bad planning phase at index %d - aborting\n", i);
      exit(1);
    }
    for(int j=0;j<sortedBucketCounts[i];j++) 
      fprintf(f, "%07d %08X\n", sortedBucketIndexes[i], pointIndexes[sortedBucketOffsets[i] + j]);
  }

  fclose(f);
  free(operations);
  return hipSuccess;   
}

