#include "hip/hip_runtime.h"
/***

Copyright (c) 2024, Snarkify, Inc. All rights reserved.
Dual licensed under the MIT License or the Apache License, Version 2.0.
See LICENSE for details.

Author(s):  Niall Emmart

***/

__global__ void array_to_montgomery_kernel(bls12381_fr* to, bls12381_fr* from, uint32_t count) {
  uint32_t tid=blockDim.x*blockIdx.x+threadIdx.x;

  if(tid>=count)
    return;

  to[tid]=bls12381_fr::to_montgomery(from[tid]);
}

__global__ void array_from_montgomery_kernel(bls12381_fr* to, bls12381_fr* from, uint32_t count) {
  uint32_t tid=blockDim.x*blockIdx.x+threadIdx.x;

  if(tid>=count)
    return;

  to[tid]=bls12381_fr::from_montgomery(from[tid]);
}

